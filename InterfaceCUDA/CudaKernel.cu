#include "hip/hip_runtime.h"
#include "CUDAKernelInterface.h"
#include <stdio.h>
#include <iostream>
//#include <fstream>


__global__ void cudaMedianFilter3x3(int width, int height, unsigned char *imgIn, unsigned char *imgOut)
{
    //int width = gridDim.x * blockDim.x;
    //int height = gridDim.y * blockDim.y;

    //__shared__ unsigned char hhh[44];

    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;

    if ((-1 < x - 1) && (-1 < y - 1) && (x + 1 < width) && (y + 1 < height))
	{
		//uchar arrayBuffer[9];
		unsigned char r0, r1, r2, r3, r4, r5, r6, r7, r8;

		const int iOffset = y * width;
		const int iPrev = iOffset - width;
		const int iNext = iOffset + width;

		//// get pixels within aperture
		r0 = imgIn[iPrev + x - 1];
		r1 = imgIn[iPrev + x];
		r2 = imgIn[iPrev + x + 1];

		r3 = imgIn[iOffset + x - 1];
		r4 = imgIn[iOffset + x];
		r5 = imgIn[iOffset + x + 1];

		r6 = imgIn[iNext + x - 1];
		r7 = imgIn[iNext + x];
		r8 = imgIn[iNext + x + 1];

		unsigned char ucharMin = min(r0, r1);
		unsigned char ucharMax = max(r0, r1);
		r0 = ucharMin;
		r1 = ucharMax;

		ucharMin = min(r3, r2);
		ucharMax = max(r3, r2);
		r3 = ucharMin;
		r2 = ucharMax;

		ucharMin = min(r2, r0);
		ucharMax = max(r2, r0);
		r2 = ucharMin;
		r0 = ucharMax;

		ucharMin = min(r3, r1);
		ucharMax = max(r3, r1);
		r3 = ucharMin;
		r1 = ucharMax;

		ucharMin = min(r1, r0);
		ucharMax = max(r1, r0);
		r1 = ucharMin;
		r0 = ucharMax;

		ucharMin = min(r3, r2);
		ucharMax = max(r3, r2);
		r3 = ucharMin;
		r2 = ucharMax;

		ucharMin = min(r5, r4);
		ucharMax = max(r5, r4);
		r5 = ucharMin;
		r4 = ucharMax;

		ucharMin = min(r7, r8);
		ucharMax = max(r7, r8);
		r7 = ucharMin;
		r8 = ucharMax;

		ucharMin = min(r6, r8);
		ucharMax = max(r6, r8);
		r6 = ucharMin;
		r8 = ucharMax;

		ucharMin = min(r6, r7);
		ucharMax = max(r6, r7);
		r6 = ucharMin;
		r7 = ucharMax;

		ucharMin = min(r4, r8);
		ucharMax = max(r4, r8);
		r4 = ucharMin;
		r8 = ucharMax;

		ucharMin = min(r4, r6);
		ucharMax = max(r4, r6);
		r4 = ucharMin;
		r6 = ucharMax;

		ucharMin = min(r5, r7);
		ucharMax = max(r5, r7);
		r5 = ucharMin;
		r7 = ucharMax;

		ucharMin = min(r4, r5);
		ucharMax = max(r4, r5);
		r4 = ucharMin;
		r5 = ucharMax;

		ucharMin = min(r6, r7);
		ucharMax = max(r6, r7);
		r6 = ucharMin;
		r7 = ucharMax;

		ucharMin = min(r0, r8);
		ucharMax = max(r0, r8);
		r0 = ucharMin;
		r8 = ucharMax;

		r4 = max(r0, r4);
		r5 = max(r1, r5);

		r6 = max(r2, r6);
		r7 = max(r3, r7);

		r4 = min(r4, r6);
		r5 = min(r5, r7);

		// store found median into result
		//result |= min(r4, r5);

		imgOut[iOffset + x] = min(r4, r5);
    }
    else if (-1 < x && x < width && -1 < y && y < height)
	{
		int centerPos = width * y + x;
		imgOut[centerPos] = imgIn[centerPos];
	}
}
void CUDAINTERFACE_MedianFilter3x3(int width, int height, unsigned char *imgIn, unsigned char *imgOut, dim3 gridSize, dim3 blockSize)
{
    //dim3 grid(4, 256); //ennyi block unk lesz
    //dim3 block(128, 2); //ennyi thread lesz egy block ban (CUDA ban egy blockhoz max 1024 thread lehet, regebbi verziokban 512)

    //int blockS;
    //int minGridS;
    //
    //hipOccupancyMaxPotentialBlockSize( &minGridS, &blockS, 
    //                                    cudaMedianFilter3x3, 0, 0); 
    //hipDeviceSynchronize(); 
    //
    //int maxActiveBlocks;
    //hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
    //                                               cudaMedianFilter3x3, blockS, 
    //                                               0);
    //int device;
    //hipDeviceProp_t props;
    //hipGetDevice(&device);
    //hipGetDeviceProperties(&props, device);
    //
    //float occupancy = (maxActiveBlocks * blockS / props.warpSize) / 
    //                (float)(props.maxThreadsPerMultiProcessor / 
    //                        props.warpSize);
    //
    //printf("Launched blocks of size %d. Theoretical occupancy: %f\n", blockS, occupancy);
    //std::cout << "Min Grid Size: ";
    //std::cout << minGridS << std::endl;
    //
    //std::cout << "Max active block Size: ";
    //std::cout << maxActiveBlocks << std::endl;
    //
    //std::cout << "Multi proc Size: ";
    //std::cout << props.maxThreadsPerMultiProcessor << std::endl;
    //
    //std::cout << "Warp Size: ";
    //std::cout << props.warpSize << std::endl;


    cudaMedianFilter3x3<<<gridSize, blockSize>>>(width, height, imgIn, imgOut);
}

__global__ void cudaCopyImage(unsigned char *imgIn, unsigned char *imgOut)
{
    int width = gridDim.x * blockDim.x;
    //int height = gridDim.y * blockDim.y;

    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;

    int pos = width * y + x;
	imgOut[pos] = imgIn[pos];
}
void CUDAINTERFACE_CopyImage(unsigned char *imgIn, unsigned char *imgOut)
{
    dim3 grid(4, 256); //ennyi block unk lesz
    dim3 block(128, 2); //ennyi thread lesz egy block ban (CUDA ban egy blockhoz max 1024 thread lehet)
    cudaCopyImage<<<grid, block>>>(imgIn, imgOut);
}

__global__ void cudaMedianFilter3x3_1D(int width, int height, unsigned char *imgIn, unsigned char *imgOut)
{
    //int width = gridDim.x * blockDim.x;
    //int height = gridDim.y * blockDim.y;

    //__shared__ unsigned char hhh[44];

    //int x = blockIdx.x*blockDim.x+threadIdx.x;
    //int y = blockIdx.y*blockDim.y+threadIdx.y;

    int x = (blockIdx.x*blockDim.x + threadIdx.x) % width;
    int y = (blockIdx.x*blockDim.x + threadIdx.x) / width;

    if ((-1 < x - 1) && (-1 < y - 1) && (x + 1 < width) && (y + 1 < height))
	{
		//uchar arrayBuffer[9];
		unsigned char r0, r1, r2, r3, r4, r5, r6, r7, r8;

		const int iOffset = y * width;
		const int iPrev = iOffset - width;
		const int iNext = iOffset + width;

		//// get pixels within aperture
		r0 = imgIn[iPrev + x - 1];
		r1 = imgIn[iPrev + x];
		r2 = imgIn[iPrev + x + 1];

		r3 = imgIn[iOffset + x - 1];
		r4 = imgIn[iOffset + x];
		r5 = imgIn[iOffset + x + 1];

		r6 = imgIn[iNext + x - 1];
		r7 = imgIn[iNext + x];
		r8 = imgIn[iNext + x + 1];

		unsigned char ucharMin = min(r0, r1);
		unsigned char ucharMax = max(r0, r1);
		r0 = ucharMin;
		r1 = ucharMax;

		ucharMin = min(r3, r2);
		ucharMax = max(r3, r2);
		r3 = ucharMin;
		r2 = ucharMax;

		ucharMin = min(r2, r0);
		ucharMax = max(r2, r0);
		r2 = ucharMin;
		r0 = ucharMax;

		ucharMin = min(r3, r1);
		ucharMax = max(r3, r1);
		r3 = ucharMin;
		r1 = ucharMax;

		ucharMin = min(r1, r0);
		ucharMax = max(r1, r0);
		r1 = ucharMin;
		r0 = ucharMax;

		ucharMin = min(r3, r2);
		ucharMax = max(r3, r2);
		r3 = ucharMin;
		r2 = ucharMax;

		ucharMin = min(r5, r4);
		ucharMax = max(r5, r4);
		r5 = ucharMin;
		r4 = ucharMax;

		ucharMin = min(r7, r8);
		ucharMax = max(r7, r8);
		r7 = ucharMin;
		r8 = ucharMax;

		ucharMin = min(r6, r8);
		ucharMax = max(r6, r8);
		r6 = ucharMin;
		r8 = ucharMax;

		ucharMin = min(r6, r7);
		ucharMax = max(r6, r7);
		r6 = ucharMin;
		r7 = ucharMax;

		ucharMin = min(r4, r8);
		ucharMax = max(r4, r8);
		r4 = ucharMin;
		r8 = ucharMax;

		ucharMin = min(r4, r6);
		ucharMax = max(r4, r6);
		r4 = ucharMin;
		r6 = ucharMax;

		ucharMin = min(r5, r7);
		ucharMax = max(r5, r7);
		r5 = ucharMin;
		r7 = ucharMax;

		ucharMin = min(r4, r5);
		ucharMax = max(r4, r5);
		r4 = ucharMin;
		r5 = ucharMax;

		ucharMin = min(r6, r7);
		ucharMax = max(r6, r7);
		r6 = ucharMin;
		r7 = ucharMax;

		ucharMin = min(r0, r8);
		ucharMax = max(r0, r8);
		r0 = ucharMin;
		r8 = ucharMax;

		r4 = max(r0, r4);
		r5 = max(r1, r5);

		r6 = max(r2, r6);
		r7 = max(r3, r7);

		r4 = min(r4, r6);
		r5 = min(r5, r7);

		// store found median into result
		//result |= min(r4, r5);

		imgOut[iOffset + x] = min(r4, r5);
    }
    else if (-1 < x && x < width && -1 < y && y < height)
	{
		int centerPos = width * y + x;
		imgOut[centerPos] = imgIn[centerPos];
	}
}
void CUDAINTERFACE_MedianFilter3x3_1D(int width, int height, unsigned char *imgIn, unsigned char *imgOut, int gridSize, int blockSize)
{
    cudaMedianFilter3x3_1D<<<gridSize, blockSize>>>(width, height, imgIn, imgOut);
}